// Copyright (c) Microsoft Corporation.
// SPDX-License-Identifier: Apache-2.0

// DeepSpeed Team

/*
Copyright NVIDIA/apex
This file is adapted from fused adam in NVIDIA/apex, commit a109f85
*/

#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/Exceptions.h>
// Another possibility:
// #include <torch/all.h>

#include <assert.h>

#include "multi_tensor_apply.cuh"
#include "type_shim.h"

#define BLOCK_SIZE 512
#define ILP 4

using MATH_T = float;

template <typename T>
struct LionFunctor {
    __device__ __forceinline__ void operator()(int chunk_size,
                                               volatile int* noop_gmem,
                                               TensorListMetadata<3>& tl,
                                               const float beta1,
                                               const float beta2,
                                               const float lr,
                                               const float decay)
    {
        // I'd like this kernel to propagate infs/nans.
        // if(*noop_gmem == 1)
        //   return;

        int tensor_loc = tl.block_to_tensor[blockIdx.x];

        // potentially use to pass in list of scalar
        // int tensor_num = tl.start_tensor_this_launch + tensor_loc;

        int chunk_idx = tl.block_to_chunk[blockIdx.x];
        int n = tl.sizes[tensor_loc];

        T* g = (T*)tl.addresses[0][tensor_loc];
        g += chunk_idx * chunk_size;

        T* p = (T*)tl.addresses[1][tensor_loc];
        p += chunk_idx * chunk_size;

        T* m = (T*)tl.addresses[2][tensor_loc];
        m += chunk_idx * chunk_size;

        n -= chunk_idx * chunk_size;

        MATH_T after_decay = 1.0f - lr * decay;

        // see note in multi_tensor_scale_kernel.cu
        for (int i_start = 0; i_start < n && i_start < chunk_size; i_start += blockDim.x * ILP) {
            MATH_T r_g[ILP];
            MATH_T r_p[ILP];
            MATH_T r_m[ILP];
#pragma unroll
            for (int ii = 0; ii < ILP; ii++) {
                int i = i_start + threadIdx.x + ii * blockDim.x;
                if (i < n && i < chunk_size) {
                    r_g[ii] = g[i];
                    r_p[ii] = p[i];
                    r_m[ii] = m[i];
                } else {
                    r_g[ii] = MATH_T(0);
                    r_p[ii] = MATH_T(0);
                    r_m[ii] = MATH_T(0);
                }
            }
#pragma unroll
            for (int ii = 0; ii < ILP; ii++) {
                MATH_T c = beta1 * r_m[ii] + (1 - beta1) * r_g[ii];
                MATH_T update = c > 0 ? (-lr) : lr;
                r_p[ii] = r_p[ii] * after_decay + update;
                r_m[ii] = beta2 * r_m[ii] + (1 - beta2) * r_g[ii];
            }
#pragma unroll
            for (int ii = 0; ii < ILP; ii++) {
                int i = i_start + threadIdx.x + ii * blockDim.x;
                if (i < n && i < chunk_size) {
                    p[i] = r_p[ii];
                    m[i] = r_m[ii];
                }
            }
        }
    }
};

void multi_tensor_lion_cuda(int chunk_size,
                            at::Tensor noop_flag,
                            std::vector<std::vector<at::Tensor>> tensor_lists,
                            const float lr,
                            const float beta1,
                            const float beta2,
                            const int step,
                            const float weight_decay)
{
    using namespace at;

    // Assume single type across p,g,m1,m2 now
    DISPATCH_DOUBLE_FLOAT_AND_HALF(tensor_lists[0][0].scalar_type(),
                                   0,
                                   "lion",
                                   multi_tensor_apply<3>(BLOCK_SIZE,
                                                         chunk_size,
                                                         noop_flag,
                                                         tensor_lists,
                                                         LionFunctor<scalar_t_0>(),
                                                         beta1,
                                                         beta2,
                                                         lr,
                                                         weight_decay);)

    AT_CUDA_CHECK(hipGetLastError());
}
