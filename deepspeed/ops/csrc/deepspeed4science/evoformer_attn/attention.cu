#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// SPDX-License-Identifier: Apache-2.0

// DeepSpeed Team

#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include "gemm_kernel_utils.h"
#include "kernel_forward.h"
#include "transform/bias_broadcast.h"

template <typename arch,
          typename scalar_t,
          typename torch_scalar_t,
          template <typename, typename, typename>
          class Broadcast1_,
          template <typename, typename, typename>
          class Broadcast2_>
typename std::enable_if<!CheckArch<arch, scalar_t>::value>::type attention_impl_template(
    torch::Tensor& q,
    torch::Tensor& k,
    torch::Tensor& v,
    torch::Tensor& bias1,
    torch::Tensor& bias2,
    torch::Tensor& o,
    float* lse_ptr)
{
    EVOFORMER_CHECK(false, "Unsupported GPU and data type combination")
}

template <typename arch,
          typename scalar_t,
          typename torch_scalar_t,
          template <typename, typename, typename>
          class Broadcast1_,
          template <typename, typename, typename>
          class Broadcast2_>
typename std::enable_if<CheckArch<arch, scalar_t>::value>::type attention_impl_template(
    torch::Tensor& q,
    torch::Tensor& k,
    torch::Tensor& v,
    torch::Tensor& bias1,
    torch::Tensor& bias2,
    torch::Tensor& o,
    float* lse_ptr)
{
    // Attention definition goes here, replaced with BroadcastType1 and
    // BroadcastType2
    using Attention = AttentionKernel<scalar_t, /* scalar_t */
                                      arch,     /* ArchTag */
                                      true,     /* Memory is aligned */
                                      64,
                                      64,
                                      true,
                                      true, /* Supports bias */
                                      Broadcast1_,
                                      Broadcast2_>;

    static_assert(!Attention::kNeedsOutputAccumulatorBuffer,
                  "This test does not support output accumulator buffer");
    int head_size = q.size(-1);
    int head_number = q.size(-2);
    int seq_length = q.size(-3);
    auto q_view = q.view({-1, seq_length, head_number, head_size});
    auto k_view = k.view({-1, seq_length, head_number, head_size});
    auto v_view = v.view({-1, seq_length, head_number, head_size});
    auto o_view = o.view({-1, seq_length, head_number, head_size});
    int batch_size = q_view.size(0);
    auto q_ptr = reinterpret_cast<scalar_t*>(q.data_ptr<torch_scalar_t>());
    auto k_ptr = reinterpret_cast<scalar_t*>(k.data_ptr<torch_scalar_t>());
    auto v_ptr = reinterpret_cast<scalar_t*>(v.data_ptr<torch_scalar_t>());
    auto o_ptr = reinterpret_cast<scalar_t*>(o.data_ptr<torch_scalar_t>());

    auto bias1_ptr = reinterpret_cast<scalar_t*>(bias1.data_ptr<torch_scalar_t>());
    auto bias2_ptr = reinterpret_cast<scalar_t*>(bias2.data_ptr<torch_scalar_t>());

    typename Attention::Params p;
    {  // set parameters
        p.query_ptr = q_ptr;
        p.key_ptr = k_ptr;
        p.value_ptr = v_ptr;
        p.logsumexp_ptr = lse_ptr;  // Only needed for bw
        p.output_accum_ptr = nullptr;
        p.output_ptr = o_ptr;
        p.scale = 1.0f / sqrt(float(head_size));

        p.bias1_ptr = bias1_ptr;
        p.bias2_ptr = bias2_ptr;
        p.B = q.size(0);
        p.N = q.size(1);

        p.num_heads = head_number;
        p.num_batches = batch_size;
        p.head_dim = head_size;
        p.head_dim_value = head_size;
        p.num_queries = seq_length;
        p.num_keys = seq_length;

        // All tensors are in BMHK shapes
        p.q_strideH = q_view.stride(-2);
        p.k_strideH = k_view.stride(-2);
        p.v_strideH = v_view.stride(-2);
        p.q_strideM = q_view.stride(-3);
        p.k_strideM = k_view.stride(-3);
        p.v_strideM = v_view.stride(-3);
        p.o_strideM = o_view.stride(-3);
        p.q_strideB = q_view.stride(-4);
        p.k_strideB = k_view.stride(-4);
        p.v_strideB = v_view.stride(-4);
    }

    constexpr auto kernel_fn = attention_kernel_batched_impl<Attention>;
    int smem_bytes = sizeof(typename Attention::SharedStorage);
    if (smem_bytes > 0xc000) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(kernel_fn), hipFuncAttributeMaxDynamicSharedMemorySize, smem_bytes);
    }
    if (!Attention::check_supported(p)) { throw std::runtime_error("Parameters not supported"); }
    kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes>>>(p);
}

#define CODE(scalar_t, torch_scalar_t)                                                          \
    do {                                                                                        \
        if (bias1.size(0) == 0 && bias2.size(0) == 0) {                                         \
            attention_impl_template<ArchTag,                                                    \
                                    scalar_t,                                                   \
                                    torch_scalar_t,                                             \
                                    BroadcastNoLoad,                                            \
                                    BroadcastNoLoad>(q, k, v, bias1, bias2, o, lse_ptr);        \
        } else if (bias1.size(0) == 0) {                                                        \
            attention_impl_template<ArchTag,                                                    \
                                    scalar_t,                                                   \
                                    torch_scalar_t,                                             \
                                    BroadcastNoLoad,                                            \
                                    BroadcastB>(q, k, v, bias1, bias2, o, lse_ptr);             \
        } else if (bias2.size(0) == 0) {                                                        \
            attention_impl_template<ArchTag,                                                    \
                                    scalar_t,                                                   \
                                    torch_scalar_t,                                             \
                                    BroadcastA,                                                 \
                                    BroadcastNoLoad>(q, k, v, bias1, bias2, o, lse_ptr);        \
        } else {                                                                                \
            attention_impl_template<ArchTag, scalar_t, torch_scalar_t, BroadcastA, BroadcastB>( \
                q, k, v, bias1, bias2, o, lse_ptr);                                             \
        }                                                                                       \
    } while (0)

// Function to select and call the correct template based on biases sizes
void attention_impl(torch::Tensor& q,
                    torch::Tensor& k,
                    torch::Tensor& v,
                    torch::Tensor& bias1,
                    torch::Tensor& bias2,
                    torch::Tensor& o,
                    torch::Tensor& lse)
{
    auto lse_ptr = lse.size(0) == 0 ? nullptr : reinterpret_cast<float*>(lse.data_ptr<float>());
    hipDeviceProp_t* prop = at::cuda::getCurrentDeviceProperties();
    DISPATCH_ARCHTAG(prop->major * 10 + prop->minor,
                     DISPATCH_TYPES(q, { CODE(scalar_t, torch_scalar_t); }));
}
