#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// SPDX-License-Identifier: Apache-2.0

// DeepSpeed Team

#include <math.h>
#include "custom_cuda_layers.h"
#include "general_kernels.h"

namespace cg = cooperative_groups;

dim3 get_attn_softmax_grid(int batch_size, int heads, int sequence_length, int threads)
{
    int seq_length4 = sequence_length / 4;
    int block_compute_size =
        (seq_length4 < threads ? (int)pow(2.0, floor(log2((float)(threads / seq_length4)))) : 1);
    // Note that the Y and Z dimensions are limited to 65535, while X is basically unlimited:
    // https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#features-and-technical-specifications
    // The batch size is typically relatively small, while the sequence length could potentially be
    // arbitrarily large. We therefore place the batch size second to avoid hitting the Y limit.
    unsigned x = heads * sequence_length / block_compute_size;
    unsigned y = batch_size;
    return {x, y};
}

// Fused attention + softmax
template <int tbSize, int blockStride, int tbSeq>
__global__ void attn_softmax(float* vals,
                             const float* attn_mask,
                             int heads,
                             int seq_length,
                             int iterations)
{
    __shared__ float partialSum[MAX_WARP_NUM];

    int warp_num = blockDim.x >> WARP_SIZE_BITS;

    int iteration_stride = blockDim.x;
    int block_width = blockStride * seq_length;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<tbSize> g = cg::tiled_partition<tbSize>(b);

    int batch = blockIdx.y;
    int row = blockIdx.x;
    int max_threads_in_sequence = std::max(seq_length, tbSeq);
    int seq_lane = threadIdx.x % max_threads_in_sequence;

    int data_offset = batch * (gridDim.x * block_width) + row * block_width +
                      (threadIdx.x / max_threads_in_sequence) * seq_length;
    int mask_offset = batch * seq_length;

    int wid = threadIdx.x >> WARP_SIZE_BITS;
    int lane = threadIdx.x & 0x1f;

    float4* val_cast = reinterpret_cast<float4*>(vals);
    const float4* attn_mask_cast = reinterpret_cast<const float4*>(attn_mask);

    float4 data[MAX_THREAD_ITERATIONS];

    float max_val = minus_infinity;

    for (int i = 0; i < iterations; i++) {
        int data_id = i * iteration_stride + seq_lane;
        if (data_id < seq_length) {
            float4 mask = attn_mask_cast[mask_offset + data_id];
            data[i] = val_cast[data_offset + data_id];

            data[i].x += mask.x;
            data[i].y += mask.y;
            data[i].z += mask.z;
            data[i].w += mask.w;

            max_val = (data[i].x > max_val ? data[i].x : max_val);
            max_val = (data[i].y > max_val ? data[i].y : max_val);
            max_val = (data[i].z > max_val ? data[i].z : max_val);
            max_val = (data[i].w > max_val ? data[i].w : max_val);
        } else {
            data[i].x = minus_infinity;
            data[i].y = minus_infinity;
            data[i].z = minus_infinity;
            data[i].w = minus_infinity;
        }
    }

    for (int i = 1; i < tbSize; i *= 2) {
        auto temp = g.shfl_xor(max_val, i);
        max_val = (temp > max_val ? temp : max_val);
    }

    if (seq_length > tbSize) {
        if (lane == 0) partialSum[wid] = max_val;
        b.sync();

        if (lane < warp_num) max_val = partialSum[lane];

#ifndef __STOCHASTIC_MODE__
        b.sync();
#endif

        int iters = warp_num;
        if (seq_length < iteration_stride)
            iters = warp_num / (iteration_stride / max_threads_in_sequence);

        for (int i = 1; i < iters; i *= 2) {
            auto temp = g.shfl_xor(max_val, i);
            max_val = (temp > max_val ? temp : max_val);
        }

        max_val = g.shfl(max_val, threadIdx.x / tbSize);
    }

    float sum = 0;
    for (int i = 0; i < iterations; i++) {
        data[i].x = __expf(data[i].x - max_val);
        data[i].y = __expf(data[i].y - max_val);
        data[i].z = __expf(data[i].z - max_val);
        data[i].w = __expf(data[i].w - max_val);

        sum += (data[i].x + data[i].y + data[i].z + data[i].w);
    }

    for (int i = 1; i < tbSize; i *= 2) { sum += g.shfl_xor(sum, i); }

    if (seq_length > tbSize) {
        if (lane == 0) partialSum[wid] = sum;
        b.sync();

        if (lane < warp_num) sum = partialSum[lane];

#ifndef __STOCHASTIC_MODE__
        b.sync();
#endif

        int iters = warp_num;
        if (seq_length < iteration_stride)
            iters = warp_num / (iteration_stride / max_threads_in_sequence);

        for (int i = 1; i < iters; i *= 2) { sum += g.shfl_xor(sum, i); }

        sum = g.shfl(sum, threadIdx.x / tbSize);
    }

    sum += 1e-6;

    for (int i = 0; i < iterations; i++) {
        data[i].x /= sum;
        data[i].y /= sum;
        data[i].z /= sum;
        data[i].w /= sum;

        int data_id = i * iteration_stride + seq_lane;
        if (data_id < seq_length) val_cast[data_offset + data_id] = data[i];
    }
}

template <int tbSize, int blockStride, int tbSeq>
__global__ void attn_softmax(__half* vals,
                             const __half* attn_mask,
                             int heads,
                             int seq_length,
                             int iterations)
{
#ifdef HALF_PRECISION_AVAILABLE
    __shared__ float partialSum[MAX_WARP_NUM];

    int warp_num = blockDim.x >> WARP_SIZE_BITS;

    int iteration_stride = blockDim.x;
    int block_width = blockStride * seq_length;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<tbSize> g = cg::tiled_partition<tbSize>(b);

    int batch = blockIdx.y;
    int row = blockIdx.x;
    int max_threads_in_sequence = std::max(seq_length, tbSeq);
    int seq_lane = threadIdx.x % max_threads_in_sequence;

    int data_offset = batch * (gridDim.x * block_width) + row * block_width +
                      (threadIdx.x / max_threads_in_sequence) * seq_length;
    int mask_offset = batch * seq_length;

    int wid = threadIdx.x >> WARP_SIZE_BITS;
    int lane = threadIdx.x & 0x1f;

    float2* val_cast = reinterpret_cast<float2*>(vals);
    const float2* attn_mask_cast = reinterpret_cast<const float2*>(attn_mask);

    val_cast += data_offset;
    attn_mask_cast += mask_offset;

    float2 low_data[MAX_THREAD_ITERATIONS];
    float2 high_data[MAX_THREAD_ITERATIONS];

    float max_val = minus_infinity;

    for (int i = 0; i < iterations; i++) {
        int data_id = i * iteration_stride + seq_lane;
        if (data_id < seq_length) {
            float2 data = val_cast[data_id];
            float2 mask = attn_mask_cast[data_id];

            __half2* data_arr = reinterpret_cast<__half2*>(&data);
            __half2* mask_arr = reinterpret_cast<__half2*>(&mask);

            low_data[i] = __half22float2(data_arr[0]);
            high_data[i] = __half22float2(data_arr[1]);
            float2 low_mask = __half22float2(mask_arr[0]);
            float2 high_mask = __half22float2(mask_arr[1]);

            low_data[i].x += low_mask.x;
            low_data[i].y += low_mask.y;
            high_data[i].x += high_mask.x;
            high_data[i].y += high_mask.y;

            max_val = (low_data[i].x > max_val ? low_data[i].x : max_val);
            max_val = (low_data[i].y > max_val ? low_data[i].y : max_val);
            max_val = (high_data[i].x > max_val ? high_data[i].x : max_val);
            max_val = (high_data[i].y > max_val ? high_data[i].y : max_val);
        }
    }

    for (int i = 1; i < tbSize; i *= 2) {
        auto temp = g.shfl_xor(max_val, i);
        max_val = (temp > max_val ? temp : max_val);
    }

    if (seq_length > tbSize) {
        if (lane == 0) partialSum[wid] = max_val;
        b.sync();

        if (lane < warp_num) max_val = partialSum[lane];

#ifndef __STOCHASTIC_MODE__
        b.sync();
#endif

        int iters = warp_num;
        if (seq_length < iteration_stride)
            iters = warp_num / (iteration_stride / max_threads_in_sequence);

        for (int i = 1; i < iters; i *= 2) {
            auto temp = g.shfl_xor(max_val, i);
            max_val = (temp > max_val ? temp : max_val);
        }

        max_val = g.shfl(max_val, threadIdx.x / tbSize);
    }

    float sum = 0;
    for (int i = 0; i < iterations; i++) {
        int data_id = i * iteration_stride + seq_lane;
        if (data_id < seq_length) {
            low_data[i].x = __expf(low_data[i].x - max_val);
            low_data[i].y = __expf(low_data[i].y - max_val);
            high_data[i].x = __expf(high_data[i].x - max_val);
            high_data[i].y = __expf(high_data[i].y - max_val);

            sum += (low_data[i].x + low_data[i].y + high_data[i].x + high_data[i].y);
        }
    }

    for (int i = 1; i < tbSize; i *= 2) { sum += g.shfl_xor(sum, i); }

    if (seq_length > tbSize) {
        if (lane == 0) partialSum[wid] = sum;
        b.sync();

        if (lane < warp_num) sum = partialSum[lane];

#ifndef __STOCHASTIC_MODE__
        b.sync();
#endif

        int iters = warp_num;
        if (seq_length < iteration_stride)
            iters = warp_num / (iteration_stride / max_threads_in_sequence);

        for (int i = 1; i < iters; i *= 2) { sum += g.shfl_xor(sum, i); }

        sum = g.shfl(sum, threadIdx.x / tbSize);
    }

    sum += 1e-6;

    for (int i = 0; i < iterations; i++) {
        int data_id = i * iteration_stride + seq_lane;
        if (data_id < seq_length) {
            float2 result_f;
            __half2* result_h = reinterpret_cast<__half2*>(&result_f);

            low_data[i].x /= sum;
            low_data[i].y /= sum;
            high_data[i].x /= sum;
            high_data[i].y /= sum;

            result_h[0] = __float22half2_rn(low_data[i]);
            result_h[1] = __float22half2_rn(high_data[i]);

            val_cast[data_id] = result_f;
        }
    }

#endif
}

template <typename T>
void launch_attn_softmax(T*, const T*, int, int, int, hipStream_t);

template <>
void launch_attn_softmax<float>(float* vals,
                                const float* attn_mask,
                                int batch_size,
                                int heads,
                                int sequence_length,
                                hipStream_t stream)
{
    const int threads = 128;
    int seq_length4 = sequence_length / 4;

    dim3 grid_dim = get_attn_softmax_grid(batch_size, heads, sequence_length, threads);

    int subblock_max_workload = MAX_THREAD_ITERATIONS * 4 * threads;

    dim3 block_dim(seq_length4 > threads ? ((sequence_length + subblock_max_workload - 1) /
                                            subblock_max_workload * threads)
                                         : threads);
    int iterations =
        (sequence_length < subblock_max_workload ? (seq_length4 + threads - 1) / threads
                                                 : MAX_THREAD_ITERATIONS);

    if (sequence_length <= 8)
        attn_softmax<2, (threads / 2), 2>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 16)
        attn_softmax<4, (threads / 4), 4>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 32)
        attn_softmax<8, (threads / 8), 8>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 64)
        attn_softmax<16, (threads / 16), 16>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 128)
        attn_softmax<32, (threads / 32), 32>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 256)
        attn_softmax<32, (threads / 64), 64>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else {
        const int threads = 256;
        dim3 grid_dim = get_attn_softmax_grid(batch_size, heads, sequence_length, threads);

        int subblock_max_workload = MAX_THREAD_ITERATIONS * 4 * threads;

        dim3 block_dim(seq_length4 > threads ? ((sequence_length + subblock_max_workload - 1) /
                                                subblock_max_workload * threads)
                                             : threads);
        iterations =
            (sequence_length < subblock_max_workload ? (seq_length4 + threads - 1) / threads
                                                     : MAX_THREAD_ITERATIONS);
        if (sequence_length <= 512)
            attn_softmax<32, (threads / 128), 128><<<grid_dim, block_dim, 0, stream>>>(
                vals, attn_mask, heads, seq_length4, iterations);
        else if (sequence_length < (MAX_THREADS * MAX_THREAD_ITERATIONS * 4))
            attn_softmax<32, 1, 128><<<grid_dim, block_dim, 0, stream>>>(
                vals, attn_mask, heads, seq_length4, iterations);
        else
            throw std::runtime_error(
                "Unsupport Seq_Length! Check the restriction of the max_threads and "
                "max_thread_iterations!");
    }
}

template <>
void launch_attn_softmax<__half>(__half* vals,
                                 const __half* attn_mask,
                                 int batch_size,
                                 int heads,
                                 int sequence_length,
                                 hipStream_t stream)
{
    const int threads = 128;
    int seq_length4 = sequence_length / 4;

    dim3 grid_dim = get_attn_softmax_grid(batch_size, heads, sequence_length, threads);

    int subblock_max_workload = MAX_THREAD_ITERATIONS * 4 * threads;

    dim3 block_dim(seq_length4 > threads ? ((sequence_length + subblock_max_workload - 1) /
                                            subblock_max_workload * threads)
                                         : threads);

    int iterations =
        (sequence_length < subblock_max_workload ? (seq_length4 + threads - 1) / threads
                                                 : MAX_THREAD_ITERATIONS);

    if (sequence_length <= 8)
        attn_softmax<2, (threads / 2), 2>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 16)
        attn_softmax<4, (threads / 4), 4>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 32)
        attn_softmax<8, (threads / 8), 8>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 64)
        attn_softmax<16, (threads / 16), 16>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 128)
        attn_softmax<32, (threads / 32), 32>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else if (sequence_length <= 256)
        attn_softmax<32, (threads / 64), 64>
            <<<grid_dim, block_dim, 0, stream>>>(vals, attn_mask, heads, seq_length4, iterations);
    else {
        const int threads = 256;
        dim3 grid_dim = get_attn_softmax_grid(batch_size, heads, sequence_length, threads);

        int subblock_max_workload = MAX_THREAD_ITERATIONS * 4 * threads;

        dim3 block_dim(seq_length4 > threads ? ((sequence_length + subblock_max_workload - 1) /
                                                subblock_max_workload * threads)
                                             : threads);
        iterations =
            (sequence_length < subblock_max_workload ? (seq_length4 + threads - 1) / threads
                                                     : MAX_THREAD_ITERATIONS);
        if (sequence_length <= 512)
            attn_softmax<32, (threads / 128), 128><<<grid_dim, block_dim, 0, stream>>>(
                vals, attn_mask, heads, seq_length4, iterations);
        else if (sequence_length < (MAX_THREADS * MAX_THREAD_ITERATIONS * 4))
            attn_softmax<32, 1, 128><<<grid_dim, block_dim, 0, stream>>>(
                vals, attn_mask, heads, seq_length4, iterations);
        else
            throw std::runtime_error(
                "Unsupport Seq_Length! Check the restriction of the max_threads and "
                "max_thread_iterations!");
    }
}

template <typename T, int tbSize, int blockStride>
__global__ void softmax_backward_kernel(T* out_grad, const T* soft_inp, int seq_length)
{
    __shared__ float partialSum[MAX_WARP_NUM];

    int warp_num = blockDim.x >> WARP_SIZE_BITS;  // warp-count = num_threads / WARP_SIZE (32)

    int iteration_stride = blockDim.x;
    int block_width = blockStride * seq_length;

    int iterations = (seq_length < (MAX_THREAD_ITERATIONS * iteration_stride)
                          ? (seq_length + iteration_stride - 1) / iteration_stride
                          : MAX_THREAD_ITERATIONS);

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<tbSize> g = cg::tiled_partition<tbSize>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;

    int wid = id >> WARP_SIZE_BITS;
    int lane = id & 0x1f;

    T val_reg[MAX_THREAD_ITERATIONS];
    T soft_reg[MAX_THREAD_ITERATIONS];
    float grad_reg = 0.0f;

#pragma unroll
    for (int i = 0; i < iterations; i++) {
        int data_id = i * iteration_stride + id;
        if (data_id < block_width) {
            val_reg[i] = out_grad[row * block_width + data_id];
            soft_reg[i] = soft_inp[row * block_width + data_id];

            grad_reg += ((float)val_reg[i] *
                         (float)soft_reg[i]);  // if done in half, the multiplication, we may lose
                                               // 2% of accuracy in computation!!
        }
    }
    for (int i = 1; i < tbSize; i *= 2) grad_reg += g.shfl_xor(grad_reg, i);

    if (seq_length > tbSize) {
        if (lane == 0) partialSum[wid] = grad_reg;
        b.sync();

        if (lane < warp_num) grad_reg = partialSum[lane];

        int iters = warp_num;
        if (seq_length < iteration_stride) iters = warp_num / (iteration_stride / seq_length);

        for (int i = 1; i < iters; i *= 2) grad_reg += g.shfl_xor(grad_reg, i);

        grad_reg = g.shfl(grad_reg, id / tbSize);
    }

    for (int i = 0; i < iterations; i++) {
        int data_id = i * iteration_stride + id;
        if (data_id < block_width) {
            float temp = (float)soft_reg[i] * ((float)val_reg[i] - grad_reg);
            out_grad[row * block_width + data_id] = (T)temp;
        }
    }
}

template <typename T, int ITERATIONS>
__global__ void softmax_backward_kernel_v2(T* grad /* input & output*/,
                                           const T* output,
                                           int softmax_length)
{
    int batch_idx = blockIdx.x * blockDim.y + threadIdx.y;
    int offset = batch_idx * softmax_length + threadIdx.x;

    grad += offset;
    output += offset;

    T grad_reg[ITERATIONS];
    T output_reg[ITERATIONS];
    float sum = 0.0;

#pragma unroll
    for (int i = 0; i < ITERATIONS; ++i) {
        int curr_idx = threadIdx.x + i * WARP_SIZE;
        if (curr_idx < softmax_length) {
            grad_reg[i] = grad[i * WARP_SIZE];
            output_reg[i] = output[i * WARP_SIZE];
            sum += (float)grad_reg[i] * (float)output_reg[i];
        }
    }

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_xor(sum, i);

#pragma unroll
    for (int i = 0; i < ITERATIONS; ++i) {
        int curr_idx = threadIdx.x + i * WARP_SIZE;
        if (curr_idx < softmax_length)
            grad[i * WARP_SIZE] = (float)output_reg[i] * ((float)grad_reg[i] - sum);
    }
}

__global__ void softmax_backward_kernel_arbitrary_length(__half* grad /* input & output*/,
                                                         const __half* output,
                                                         int softmax_length)
{
    int batch_idx = blockIdx.x * blockDim.y + threadIdx.y;
    int offset = batch_idx * softmax_length + threadIdx.x;

    const float4* output_cast = reinterpret_cast<const float4*>(output);
    float4* grad_cast = reinterpret_cast<float4*>(grad);

    grad_cast += offset;
    output_cast += offset;

    float sum = 0.0;
    int curr_idx = threadIdx.x;
    while (curr_idx < softmax_length) {
        float4 out_reg = output_cast[curr_idx];
        float4 grad_reg = grad_cast[curr_idx];
        __half2* out_h = reinterpret_cast<__half2*>(&out_reg);
        __half2* grad_h = reinterpret_cast<__half2*>(&grad_reg);
#pragma unroll
        for (int m = 0; m < 4; m++) grad_h[m] *= out_h[m];
        sum += ((float)grad_h[0].x + (float)grad_h[0].y + (float)grad_h[1].x + (float)grad_h[1].y) +
               ((float)grad_h[2].x + (float)grad_h[2].y + (float)grad_h[3].x + (float)grad_h[3].y);
        curr_idx += WARP_SIZE;
    }

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

#pragma unroll
    for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_xor(sum, i);

    curr_idx = threadIdx.x;
    while (curr_idx < softmax_length) {
        float4 out_reg = output_cast[curr_idx];
        float4 grad_reg = grad_cast[curr_idx];
        __half* grad_h = reinterpret_cast<__half*>(&grad_reg);
        __half* out_h = reinterpret_cast<__half*>(&out_reg);

#pragma unroll
        for (int m = 0; m < 8; m++) grad_h[m] = (float)out_h[m] * ((float)grad_h[m] - sum);

        grad_cast[curr_idx] = grad_reg;
        curr_idx += WARP_SIZE;
    }
}

__global__ void softmax_backward_kernel_arbitrary_length(float* grad /* input & output*/,
                                                         const float* output,
                                                         int softmax_length)
{
    int batch_idx = blockIdx.x * blockDim.y + threadIdx.y;
    int offset = batch_idx * softmax_length + threadIdx.x;

    const float4* output_cast = reinterpret_cast<const float4*>(output);
    float4* grad_cast = reinterpret_cast<float4*>(grad);

    grad_cast += offset;
    output_cast += offset;

    float sum = 0.0;
    int curr_idx = threadIdx.x;
    while (curr_idx < softmax_length) {
        float4 out_reg = output_cast[curr_idx];
        float4 grad_reg = grad_cast[curr_idx];

        grad_reg.x *= out_reg.x;
        grad_reg.y *= out_reg.y;
        grad_reg.z *= out_reg.z;
        grad_reg.w *= out_reg.w;
        sum += (grad_reg.x + grad_reg.y + grad_reg.z + grad_reg.w);

        curr_idx += WARP_SIZE;
    }

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

#pragma unroll
    for (int i = 1; i < WARP_SIZE; i <<= 1) sum += g.shfl_xor(sum, i);

    curr_idx = threadIdx.x;
    while (curr_idx < softmax_length) {
        float4 out_reg = output_cast[curr_idx];
        float4 grad_reg = grad_cast[curr_idx];
        grad_reg.x = out_reg.x * (grad_reg.x - sum);
        grad_reg.y = out_reg.y * (grad_reg.y - sum);
        grad_reg.z = out_reg.z * (grad_reg.z - sum);
        grad_reg.w = out_reg.w * (grad_reg.w - sum);

        grad_cast[curr_idx] = grad_reg;
        curr_idx += WARP_SIZE;
    }
}

template <typename T>
void launch_attn_softmax_backward_v2(T* out_grad,
                                     const T* soft_inp,
                                     int batch_size,
                                     int heads,
                                     int seq_length,
                                     hipStream_t stream)
{
    const int warps_per_block = 4;
    dim3 grid_dim(batch_size * heads * seq_length / warps_per_block);
    dim3 block_dim(WARP_SIZE, warps_per_block);

    if (seq_length <= 32)
        softmax_backward_kernel_v2<T, 1>
            <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
    else if (seq_length <= 64)
        softmax_backward_kernel_v2<T, 2>
            <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
    else if (seq_length <= 128)
        softmax_backward_kernel_v2<T, 4>
            <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
    else if (seq_length <= 256)
        softmax_backward_kernel_v2<T, 8>
            <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
    else if (seq_length <= 384)
        softmax_backward_kernel_v2<T, 12>
            <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
    else if (seq_length <= 512)
        softmax_backward_kernel_v2<T, 16>
            <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
    else if (seq_length <= 768)
        softmax_backward_kernel_v2<T, 24>
            <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
    else if (seq_length <= 1024)
        softmax_backward_kernel_v2<T, 32>
            <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
    else if (seq_length <= 2048)
        softmax_backward_kernel_v2<T, 64>
            <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
    else if (seq_length <= 4096)
        softmax_backward_kernel_v2<T, 128>
            <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
    else if (seq_length <= 8192)
        softmax_backward_kernel_v2<T, 256>
            <<<grid_dim, block_dim, 0, stream>>>(out_grad, soft_inp, seq_length);
    else
        softmax_backward_kernel_arbitrary_length<<<grid_dim, block_dim, 0, stream>>>(
            out_grad, soft_inp, seq_length / (4 << ((sizeof(T) & 2) >> 1)));
}

template void launch_attn_softmax_backward_v2<__half>(__half* out_grad,
                                                      const __half* soft_inp,
                                                      int batch_size,
                                                      int heads,
                                                      int seq_length,
                                                      hipStream_t stream);
template void launch_attn_softmax_backward_v2<float>(float* out_grad,
                                                     const float* soft_inp,
                                                     int batch_size,
                                                     int heads,
                                                     int seq_length,
                                                     hipStream_t stream);
