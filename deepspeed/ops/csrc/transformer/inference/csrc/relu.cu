#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// SPDX-License-Identifier: Apache-2.0

// DeepSpeed Team

#include "conversion_utils.h"
#include "inference_cuda_layers.h"
#include "memory_access_utils.h"

namespace cg = cooperative_groups;
#define MAX_CAP 4
#define MAX_SEQ 2048

inline __device__ float relu(const float x) { return x < 0 ? 0 : x; }

/*
In-place relu(biasAdd(x)) for channels last
*/
template <typename T>
__global__ void fused_bias_relu(T* input, const T* bias, int total_count, int intermediate_size)
{
    // Input restriction: intermediate_size % vals_per_access == 0
    constexpr int granularity = 16;
    constexpr int values_per_access = granularity / sizeof(T);
    const int offset = (blockIdx.x * blockDim.x + threadIdx.x) * values_per_access;

    if (offset < total_count) {
        T data[values_per_access];
        T data_bias[values_per_access];
        mem_access::load_global<granularity>(data, input + offset);
        mem_access::load_global<granularity>(
            data_bias, bias + (offset % intermediate_size), bias != nullptr);

#pragma unroll
        for (int i = 0; i < values_per_access; i++) {
            float data_f = conversion::to<float>(data[i]);
            float bias_f = conversion::to<float>(data_bias[i]);
            data[i] = conversion::to<T>(relu(data_f + bias_f));
        }

        mem_access::store_global<granularity>(input + offset, data);
    }
}

template <typename T>
void launch_bias_relu(T* input,
                      const T* bias,
                      int intermediate_size,
                      int batch_size,
                      hipStream_t stream)
{
    constexpr int threads = 1024;
    constexpr int granularity = 16;

    const int total_count = batch_size * intermediate_size;
    const int elems_per_block = threads * (granularity / sizeof(T));
    dim3 block_dims(threads);
    dim3 grid_dims((total_count + elems_per_block - 1) / elems_per_block);

    fused_bias_relu<<<grid_dims, block_dims, 0, stream>>>(
        input, bias, total_count, intermediate_size);
}

#define INSTANTIATE_LAUNCH_BIAS_RELU(T) \
    template void launch_bias_relu<T>(T*, const T*, int, int, hipStream_t);

INSTANTIATE_LAUNCH_BIAS_RELU(float)
#ifdef BF16_AVAILABLE
INSTANTIATE_LAUNCH_BIAS_RELU(__hip_bfloat16)
#endif
INSTANTIATE_LAUNCH_BIAS_RELU(__half)
