#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// SPDX-License-Identifier: Apache-2.0

// DeepSpeed Team

#include "conversion_utils.h"
#include "inference_cuda_layers.h"

#define MAX_QUANTIZE_GROUPING 1024

#define loop_unroll 1
#define loop_unroll_bits 1

template <typename T>
__global__ void dequantize_kernel(T* output,
                                  const int8_t* input,
                                  const float* qscale,
                                  int output_size,
                                  int hidden_dim,
                                  int groups,
                                  int merge_count)
{
    unsigned merge_hidden = hidden_dim >> merge_count;
    unsigned quantization_stride = (merge_hidden * output_size) / groups;

    unsigned bid = blockIdx.x;
    unsigned tid = threadIdx.x;

    while (tid < output_size) {
        unsigned w_index = bid / merge_hidden;
        unsigned q_index = tid + bid * output_size;

        auto q = input[q_index];

        unsigned merge_hidden_total = w_index * merge_hidden;
        unsigned scale_index =
            ((((bid - merge_hidden_total) + tid * merge_hidden) / quantization_stride)
             << merge_count) +
            w_index;

        float scale_data = qscale[scale_index];

        output[q_index] = conversion::to<T>(scale_data * (float)q);
        tid += blockDim.x;
    }
}

template <typename T>
void launch_dequantize(T* output,
                       const int8_t* input,
                       const float* qscale,
                       unsigned output_size,
                       unsigned hidden_dim,
                       unsigned groups,
                       unsigned merge_count,
                       hipStream_t stream)
{
    unsigned threads = 1024;
    dim3 block_dims(threads);
    dim3 grid_dims(hidden_dim);

    dequantize_kernel<<<grid_dims, block_dims, 0, stream>>>(
        output, input, qscale, output_size, hidden_dim, groups, merge_count);
}

#define INSTANTIATE_DEQUANTIZE_MERGE(T) \
    template void launch_dequantize<T>( \
        T*, const int8_t*, const float*, unsigned, unsigned, unsigned, unsigned, hipStream_t);

INSTANTIATE_DEQUANTIZE_MERGE(float);
#ifdef BF16_AVAILABLE
INSTANTIATE_DEQUANTIZE_MERGE(__hip_bfloat16);
#endif
INSTANTIATE_DEQUANTIZE_MERGE(__half);

__global__ void dequantize_kernel(float* output,
                                  const int8_t* input,
                                  const float* qscale,
                                  int hidden_dim,
                                  unsigned merge_hidden,
                                  int cnt)
{
}

template <typename T>
__global__ void dequantize_kernel(T* output,
                                  const int8_t* input,
                                  const float* qscale,
                                  unsigned hidden_dim,
                                  unsigned merge_hidden,
                                  int cnt)
{
    unsigned bid = blockIdx.x * gridDim.y + blockIdx.y;
    unsigned tid = threadIdx.x;

    float local_scale = qscale[blockIdx.x];

    const float* input_cast = reinterpret_cast<const float*>(input);
    float2* output_cast = reinterpret_cast<float2*>(output);

    input_cast += bid * merge_hidden;
    output_cast += bid * merge_hidden;

    for (int c = 0; c < cnt; c++) {
        if (tid < merge_hidden) {
            float q = input_cast[tid];
            int8_t* q_int8 = (int8_t*)&q;

            float2 q_f;
            T* q_h = (T*)&q_f;

            q_h[0] = conversion::to<T>(local_scale * (float)q_int8[0]);
            q_h[1] = conversion::to<T>(local_scale * (float)q_int8[1]);
            q_h[2] = conversion::to<T>(local_scale * (float)q_int8[2]);
            q_h[3] = conversion::to<T>(local_scale * (float)q_int8[3]);
            output_cast[tid] = q_f;
            tid += blockDim.x;
        }
    }
}

template <typename T>
void launch_dequantize(T* output,
                       const int8_t* input,
                       const float* qscale,
                       unsigned output_size,
                       unsigned hidden_dim,
                       unsigned groups,
                       hipStream_t stream)
{
    unsigned threads = 1024;
    hidden_dim /= 4;
    unsigned hid_cnt = threads / hidden_dim;
    unsigned thd_cnt = (hidden_dim - 1) / threads + 1;
    hid_cnt = hid_cnt > 0 ? hid_cnt : 1;

    unsigned blocks = (output_size + hid_cnt * groups - 1) / (hid_cnt * groups);
    dim3 block_dims(threads);
    dim3 grid_dims(groups, blocks);

    dequantize_kernel<<<grid_dims, block_dims, 0, stream>>>(
        output, input, qscale, hidden_dim, hid_cnt * hidden_dim, thd_cnt);
}

#define INSTANTIATE_DEQUANTIZE_NO_MERGE(T) \
    template void launch_dequantize<T>(    \
        T*, const int8_t*, const float*, unsigned, unsigned, unsigned, hipStream_t);

INSTANTIATE_DEQUANTIZE_NO_MERGE(float);
#ifdef BF16_AVAILABLE
INSTANTIATE_DEQUANTIZE_NO_MERGE(__hip_bfloat16);
#endif
INSTANTIATE_DEQUANTIZE_NO_MERGE(__half);
