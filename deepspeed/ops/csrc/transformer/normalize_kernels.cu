#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// SPDX-License-Identifier: Apache-2.0

// DeepSpeed Team

#include "custom_cuda_layers.h"

namespace cg = cooperative_groups;

/*
Fused bias add, residual (elementwise) add, and normalization layer.

For FP16, this kernel does not promote to FP32 in order to utilize the 2x throughput for
__half2 instructions, and avoid the conversion overhead (1/8 of __hal2 arithmetic).

For specific launch constraints, see the launch functions.
*/

#define NORM_REG (MAX_REGISTERS / 4)

__global__ void fused_bias_residual_layer_norm(float* vals,
                                               const float* residual,
                                               const float* gamma,
                                               const float* beta,
                                               float epsilon,
                                               bool preLayerNorm,
                                               bool training,
                                               float* vars,
                                               float* means,
                                               int row_stride)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int gid = id / WARP_SIZE;

    float vals_arr[NORM_REG];
    __shared__ float shr[MAX_WARP_NUM];

    residual += (row * row_stride);
    vals += (row * row_stride);

    float sum = 0.f;
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] = residual[i * iteration_stride + id];
        sum += vals_arr[i];
    }
    if (high_index < row_stride) {
        vals_arr[iterations] = residual[high_index];
        sum += vals_arr[iterations];
        iterations++;
    }

    for (int i = 1; i < 32; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) shr[gid] = sum;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> WARP_SIZE_BITS)) sum = shr[g.thread_rank()];

#if !defined(__STOCHASTIC_MODE__) || __CUDA_ARCH__ < 700
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> WARP_SIZE_BITS); i *= 2) {
        sum += g.shfl_down(sum, i);
    }

    sum = g.shfl(sum, 0);
    float mean = sum / row_stride;
    if (training)
        if (threadIdx.x == 0) means[row] = mean;
    float variance = 0.f;
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] -= mean;
        variance += vals_arr[i] * vals_arr[i];
    }

    for (int i = 1; i < 32; i *= 2) { variance += g.shfl_down(variance, i); }

    if (g.thread_rank() == 0) shr[gid] = variance;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> WARP_SIZE_BITS)) variance = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> WARP_SIZE_BITS); i *= 2) {
        variance += g.shfl_down(variance, i);
    }
    variance = g.shfl(variance, 0);
    variance /= row_stride;
    variance += epsilon;
    if (training)
        if (threadIdx.x == 0) vars[row] = variance;

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] = vals_arr[i] * rsqrtf(variance);
        vals_arr[i] =
            vals_arr[i] * gamma[i * iteration_stride + id] + beta[i * iteration_stride + id];
        vals[i * iteration_stride + id] = vals_arr[i];
    }
    if ((high_index) < row_stride) {
        vals_arr[iterations] = vals_arr[iterations] * rsqrtf(variance);
        vals_arr[iterations] = vals_arr[iterations] * gamma[high_index] + beta[high_index];
        vals[high_index] = vals_arr[iterations];
    }
}

__global__ void fused_bias_residual_layer_norm(__half* vals,
                                               const __half* residual,
                                               const __half* gamma,
                                               const __half* beta,
                                               float epsilon,
                                               bool preLayerNorm,
                                               bool training,
                                               __half* vars,
                                               __half* means,
                                               int row_stride)
{
#ifdef HALF_PRECISION_AVAILABLE
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<32> g = cg::tiled_partition<32>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int gid = id >> WARP_SIZE_BITS;

    float2 vals_f[NORM_REG];
    __shared__ float shr[MAX_WARP_NUM];

    __half2* vals_cast = reinterpret_cast<__half2*>(vals);
    const __half2* residual_cast = reinterpret_cast<const __half2*>(residual);

    residual_cast += (row * row_stride);
    vals_cast += (row * row_stride);

    float sum = 0.f;
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        vals_f[i] = __half22float2(residual_cast[i * iteration_stride + id]);
        sum += vals_f[i].x;
        sum += vals_f[i].y;
    }
    if ((high_index) < row_stride) {
        vals_f[iterations] = __half22float2(residual_cast[high_index]);
        sum += vals_f[iterations].x;
        sum += vals_f[iterations].y;
        iterations++;
    }

    for (int i = 1; i < 32; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) shr[gid] = sum;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> WARP_SIZE_BITS)) sum = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> WARP_SIZE_BITS); i *= 2) {
        sum += g.shfl_down(sum, i);
    }
    sum = g.shfl(sum, 0);
    float mean = sum / (row_stride * 2);

    float variance = 0.f;
    for (int i = 0; i < iterations; i++) {
        vals_f[i].x -= mean;
        vals_f[i].y -= mean;
        variance += vals_f[i].x * vals_f[i].x;
        variance += vals_f[i].y * vals_f[i].y;
    }

    for (int i = 1; i < 32; i *= 2) { variance += g.shfl_down(variance, i); }

    if (g.thread_rank() == 0) shr[gid] = variance;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> WARP_SIZE_BITS)) variance = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> WARP_SIZE_BITS); i *= 2) {
        variance += g.shfl_down(variance, i);
    }
    variance = g.shfl(variance, 0);
    variance /= (row_stride * 2);
    variance += epsilon;

    __half2 variance_h = __float2half2_rn(variance);
    const __half2* gamma_cast = reinterpret_cast<const __half2*>(gamma);
    const __half2* beta_cast = reinterpret_cast<const __half2*>(beta);

    if (training && threadIdx.x == 0) {
        vars[row] = __float2half(variance);
        means[row] = __float2half(mean);
    }
    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        __half2 vals_arr = __float22half2_rn(vals_f[i]);
        vals_arr = vals_arr * h2rsqrt(variance_h);
        vals_arr =
            vals_arr * gamma_cast[i * iteration_stride + id] + beta_cast[i * iteration_stride + id];
        vals_cast[i * iteration_stride + id] = vals_arr;
    }
    if ((high_index) < row_stride) {
        __half2 vals_arr = __float22half2_rn(vals_f[iterations]);
        vals_arr = vals_arr * h2rsqrt(variance_h);
        vals_arr = vals_arr * gamma_cast[high_index] + beta_cast[high_index];
        vals_cast[high_index] = vals_arr;
    }
#endif
}

template <typename T>
void launch_bias_residual_layer_norm(T* vals,
                                     const T* residual,
                                     const T* gamma,
                                     const T* beta,
                                     float epsilon,
                                     int batch_size,
                                     int hidden_dim,
                                     hipStream_t stream,
                                     bool preLayerNorm,
                                     bool training,
                                     T* vars,
                                     T* means);

template <>
void launch_bias_residual_layer_norm<float>(float* vals,
                                            const float* residual,
                                            const float* gamma,
                                            const float* beta,
                                            float epsilon,
                                            int batch_size,
                                            int hidden_dim,
                                            hipStream_t stream,
                                            bool preLayerNorm,
                                            bool training,
                                            float* vars,
                                            float* means)
{
    int threads = THREADS;

    dim3 grid_dim(batch_size);

    if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 1;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 2;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim(threads);

    fused_bias_residual_layer_norm<<<grid_dim, block_dim, 0, stream>>>(
        vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means, hidden_dim);
}

template <>
void launch_bias_residual_layer_norm<__half>(__half* vals,
                                             const __half* residual,
                                             const __half* gamma,
                                             const __half* beta,
                                             float epsilon,
                                             int batch_size,
                                             int hidden_dim,
                                             hipStream_t stream,
                                             bool preLayerNorm,
                                             bool training,
                                             __half* vars,
                                             __half* means)
{
    int threads = 128;

    dim3 grid_dim(batch_size);

    if (hidden_dim > 8192 && hidden_dim <= 16384)
        threads <<= 1;
    else if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 2;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 3;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim(threads);

    fused_bias_residual_layer_norm<<<grid_dim, block_dim, 0, stream>>>(
        vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, means, hidden_dim / 2);
}

__global__ void fused_bias_residual_layer_norm(float* vals,
                                               const float* residual,
                                               const float* gamma,
                                               const float* beta,
                                               float epsilon,
                                               bool preLayerNorm,
                                               bool training,
                                               float* vars,
                                               int row_stride)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<32> g = cg::tiled_partition<32>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int gid = id / 32;

    float vals_arr[NORM_REG];
    __shared__ float shr[MAX_WARP_NUM];

    residual += (row * row_stride);
    vals += (row * row_stride);

    float sum = 0.f;
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] = residual[i * iteration_stride + id];
        sum += vals_arr[i];
    }
    if ((high_index) < row_stride) {
        vals_arr[iterations] = residual[high_index];
        sum += vals_arr[iterations];
        iterations++;
    }

    for (int i = 1; i < 32; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) shr[gid] = sum;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> WARP_SIZE_BITS)) sum = shr[g.thread_rank()];

#if !defined(__STOCHASTIC_MODE__) || __CUDA_ARCH__ < 700
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> WARP_SIZE_BITS); i *= 2) {
        sum += g.shfl_down(sum, i);
    }

    sum = g.shfl(sum, 0);
    float mean = sum / row_stride;
    float variance = 0.f;
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] -= mean;
        variance += vals_arr[i] * vals_arr[i];
    }

    for (int i = 1; i < 32; i *= 2) { variance += g.shfl_down(variance, i); }

    if (g.thread_rank() == 0) shr[gid] = variance;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> WARP_SIZE_BITS)) variance = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> WARP_SIZE_BITS); i *= 2) {
        variance += g.shfl_down(variance, i);
    }
    variance = g.shfl(variance, 0);
    variance /= row_stride;
    variance += epsilon;
    if (training)
        if (threadIdx.x == 0) vars[row] = variance;

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        vals_arr[i] = vals_arr[i] * rsqrtf(variance);
        vals_arr[i] =
            vals_arr[i] * gamma[i * iteration_stride + id] + beta[i * iteration_stride + id];
        vals[i * iteration_stride + id] = vals_arr[i];
    }
    if ((high_index) < row_stride) {
        vals_arr[iterations] = vals_arr[iterations] * rsqrtf(variance);
        vals_arr[iterations] = vals_arr[iterations] * gamma[high_index] + beta[high_index];
        vals[high_index] = vals_arr[iterations];
    }
}

__global__ void fused_bias_residual_layer_norm(__half* vals,
                                               const __half* residual,
                                               const __half* gamma,
                                               const __half* beta,
                                               float epsilon,
                                               bool preLayerNorm,
                                               bool training,
                                               __half* vars,
                                               int row_stride)
{
#ifdef HALF_PRECISION_AVAILABLE

    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<32> g = cg::tiled_partition<32>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int gid = id >> WARP_SIZE_BITS;

    float2 vals_f[NORM_REG];
    __shared__ float shr[MAX_WARP_NUM];

    __half2* vals_cast = reinterpret_cast<__half2*>(vals);
    const __half2* residual_cast = reinterpret_cast<const __half2*>(residual);

    residual_cast += (row * row_stride);
    vals_cast += (row * row_stride);

    float sum = 0.f;
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        vals_f[i] = __half22float2(residual_cast[i * iteration_stride + id]);
        sum += vals_f[i].x;
        sum += vals_f[i].y;
    }
    if ((high_index) < row_stride) {
        vals_f[iterations] = __half22float2(residual_cast[high_index]);
        sum += vals_f[iterations].x;
        sum += vals_f[iterations].y;
        iterations++;
    }

    for (int i = 1; i < 32; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) shr[gid] = sum;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> WARP_SIZE_BITS)) sum = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> WARP_SIZE_BITS); i *= 2) {
        sum += g.shfl_down(sum, i);
    }
    sum = g.shfl(sum, 0);
    float mean = sum / (row_stride * 2);

    float variance = 0.f;
    for (int i = 0; i < iterations; i++) {
        vals_f[i].x -= mean;
        vals_f[i].y -= mean;
        variance += vals_f[i].x * vals_f[i].x;
        variance += vals_f[i].y * vals_f[i].y;
    }

    for (int i = 1; i < 32; i *= 2) { variance += g.shfl_down(variance, i); }

    if (g.thread_rank() == 0) shr[gid] = variance;

    b.sync();

    if (g.thread_rank() < (iteration_stride >> WARP_SIZE_BITS)) variance = shr[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    b.sync();
#endif

    for (int i = 1; i < (iteration_stride >> WARP_SIZE_BITS); i *= 2) {
        variance += g.shfl_down(variance, i);
    }
    variance = g.shfl(variance, 0);
    variance /= (row_stride * 2);
    variance += epsilon;

    __half2 variance_h = __float2half2_rn(variance);
    const __half2* gamma_cast = reinterpret_cast<const __half2*>(gamma);
    const __half2* beta_cast = reinterpret_cast<const __half2*>(beta);

    if (training && threadIdx.x == 0) vars[row] = __float2half(variance);

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        __half2 vals_arr = __float22half2_rn(vals_f[i]);
        vals_arr = vals_arr * h2rsqrt(variance_h);
        vals_arr =
            vals_arr * gamma_cast[i * iteration_stride + id] + beta_cast[i * iteration_stride + id];
        vals_cast[i * iteration_stride + id] = vals_arr;
    }
    if ((high_index) < row_stride) {
        __half2 vals_arr = __float22half2_rn(vals_f[iterations]);
        vals_arr = vals_arr * h2rsqrt(variance_h);
        vals_arr = vals_arr * gamma_cast[high_index] + beta_cast[high_index];
        vals_cast[high_index] = vals_arr;
    }
#endif
}

template <typename T>
void launch_bias_residual_layer_norm(T* vals,
                                     const T* residual,
                                     const T* gamma,
                                     const T* beta,
                                     float epsilon,
                                     int batch_size,
                                     int hidden_dim,
                                     hipStream_t stream,
                                     bool preLayerNorm,
                                     bool training,
                                     T* vars);

/*
To tune this launch the following restrictions must be met:

For float:
row_stride == hidden_size
threads * iterations == row_stride
threads is in [32, 64, 128, 256, 512, 1024]

For half:
row_stride == hidden_size / 2
threads * iterations == row_stride
threads is in [32, 64, 128, 256, 512, 1024]

*/

template <>
void launch_bias_residual_layer_norm<float>(float* vals,
                                            const float* residual,
                                            const float* gamma,
                                            const float* beta,
                                            float epsilon,
                                            int batch_size,
                                            int hidden_dim,
                                            hipStream_t stream,
                                            bool preLayerNorm,
                                            bool training,
                                            float* vars)
{
    int threads = THREADS;

    dim3 grid_dim(batch_size);

    // There are some limitations to call below functions, now just enumerate the situations.

    if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 1;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 2;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim(threads);

    fused_bias_residual_layer_norm<<<grid_dim, block_dim, 0, stream>>>(
        vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, hidden_dim);
}

template <>
void launch_bias_residual_layer_norm<__half>(__half* vals,
                                             const __half* residual,
                                             const __half* gamma,
                                             const __half* beta,
                                             float epsilon,
                                             int batch_size,
                                             int hidden_dim,
                                             hipStream_t stream,
                                             bool preLayerNorm,
                                             bool training,
                                             __half* vars)
{
    int threads = 128;

    dim3 grid_dim(batch_size);

    // There are some limitations to call below functions, now just enumerate the situations.

    if (hidden_dim > 8192 && hidden_dim <= 16384)
        threads <<= 1;
    else if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 2;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 3;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim(threads);
    fused_bias_residual_layer_norm<<<grid_dim, block_dim, 0, stream>>>(
        vals, residual, gamma, beta, epsilon, preLayerNorm, training, vars, hidden_dim / 2);
}

/* Normalize Gamma & Betta gradients
 * Compute gradients using either X_hat or
 * normalize input (invertible).
 * Combine transpose with gradients computation.
 */

template <typename T>
__global__ void LayerNormBackward1(const T* __restrict__ out_grad,
                                   const T* __restrict__ vals_hat,
                                   const T* __restrict__ gamma,
                                   const T* __restrict__ betta,
                                   T* __restrict__ gamma_grad,
                                   T* __restrict__ betta_grad,
                                   int rows,
                                   int width,
                                   bool invertible)
{
    __shared__ float betta_buffer[TILE_DIM][TILE_DIM + 1];
    __shared__ float gamma_buffer[TILE_DIM][TILE_DIM + 1];

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = threadIdx.y * width + idx;
    int y_stride = width * TILE_DIM;

    float betta_reg = (invertible ? (float)betta[idx] : 0.0f);
    float gamma_reg = (float)gamma[idx];

    // Loop across matrix height
    float betta_tmp = 0;
    float gamma_tmp = 0;
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        float grad = (float)out_grad[offset];
        float val = (invertible ? ((float)vals_hat[offset] - betta_reg) / gamma_reg
                                : (float)vals_hat[offset]);
        betta_tmp += grad;
        gamma_tmp += (val * grad);

        offset += y_stride;
    }

    betta_buffer[threadIdx.x][threadIdx.y] = betta_tmp;
    gamma_buffer[threadIdx.x][threadIdx.y] = gamma_tmp;

    __syncthreads();

    // Sum the shared buffer.
    float s1 = betta_buffer[threadIdx.y][threadIdx.x];
    float s2 = gamma_buffer[threadIdx.y][threadIdx.x];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < TILE_DIM; i <<= 1) {
        s1 += g.shfl_down(s1, i);
        s2 += g.shfl_down(s2, i);
    }

    if (threadIdx.x == 0) {
        int pos = blockIdx.x * TILE_DIM + threadIdx.y;
        betta_grad[pos] = s1;
        gamma_grad[pos] = s2;
    }
}

/* Normalize Gamma & Betta gradients
 * Compute gradients using the input to
 * the normalize.
 * Combine transpose with gradients computation.
 */

template <typename T>
__global__ void LayerNormBackward1(const T* __restrict__ out_grad,
                                   const T* __restrict__ X_data,
                                   const T* __restrict__ vars,
                                   const T* __restrict__ means,
                                   T* __restrict__ gamma_grad,
                                   T* __restrict__ betta_grad,
                                   int rows,
                                   int width)
{
    __shared__ float betta_buffer[TILE_DIM][TILE_DIM + 1];
    __shared__ float gamma_buffer[TILE_DIM][TILE_DIM + 1];

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = threadIdx.y * width + idx;
    int y_stride = width * TILE_DIM;

    int pos = blockIdx.x * TILE_DIM + threadIdx.y;
    // Loop across matrix height

    float betta_tmp = 0;
    float gamma_tmp = 0;
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        float grad = (float)out_grad[offset];
        float val = (float)X_data[offset];
        val = (val - (float)means[r]) * rsqrtf((float)vars[r]);
        betta_tmp += grad;
        gamma_tmp += (val * grad);

        offset += y_stride;
    }

    betta_buffer[threadIdx.x][threadIdx.y] = betta_tmp;
    gamma_buffer[threadIdx.x][threadIdx.y] = gamma_tmp;

    __syncthreads();

    // Sum the shared buffer.
    float s1 = betta_buffer[threadIdx.y][threadIdx.x];
    float s2 = gamma_buffer[threadIdx.y][threadIdx.x];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < TILE_DIM; i <<= 1) {
        s1 += g.shfl_down(s1, i);
        s2 += g.shfl_down(s2, i);
    }

    if (threadIdx.x == 0) {
        betta_grad[pos] = s1;
        gamma_grad[pos] = s2;
    }
}
/*

/* Backward Normalize (Input-Gradient)
 * Using the means and variances from the input
 * This type of backward is invertible!
 * We do the backward using the X_hat (X - u) / sqrt(variance) or the output of Normalization.
 */

__global__ void LayerNormBackward2(const float* out_grad,
                                   const float* vals_hat,
                                   const float* gamma,
                                   const float* betta,
                                   const float* vars,
                                   float* inp_grad,
                                   bool invertible,
                                   int row_stride)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    int warp_num = iteration_stride >> WARP_SIZE_BITS;
    __shared__ float partialSum[MAX_WARP_NUM];

    out_grad += (row * row_stride);
    vals_hat += (row * row_stride);
    inp_grad += (row * row_stride);

    float vals_arr[NORM_REG];
    float vals_hat_arr[NORM_REG];
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        float gamma_reg = gamma[i * iteration_stride + id];
        vals_arr[i] = out_grad[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
        vals_hat_arr[i] =
            (invertible ? (vals_hat[i * iteration_stride + id] - betta[i * iteration_stride + id]) /
                              gamma_reg
                        : vals_hat[i * iteration_stride + id]);
    }
    if ((high_index) < row_stride) {
        float gamma_reg = gamma[high_index];
        vals_arr[iterations] = out_grad[high_index];
        vals_arr[iterations] *= gamma_reg;
        vals_hat_arr[iterations] =
            (invertible ? (vals_hat[high_index] - betta[high_index]) / gamma_reg
                        : vals_hat[high_index]);
        iterations++;
    }

    float var_reg = vars[row];

    float sum = 0;
    for (int i = 0; i < iterations; i++) {
        sum += vals_hat_arr[i] * vals_arr[i] *
               sqrtf(var_reg);           // dval_hat = gamma * (x - u) * out_grad
        vals_arr[i] *= rsqrtf(var_reg);  // dvar_inv = gamma * out_grad / sqrt(var)
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++) { vals_arr[i] += ((-sum * vals_hat_arr[i]) / var_reg); }

    sum = 0;
    for (int i = 0; i < iterations; i++) { sum += vals_arr[i]; }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);
    sum = g.shfl(sum, 0);
    sum /= row_stride;

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) inp_grad[i * iteration_stride + id] = (vals_arr[i] - sum);
    if ((high_index) < row_stride) inp_grad[high_index] = (vals_arr[iterations] - sum);
}

__global__ void LayerNormBackward2(const __half* out_grad,
                                   const __half* vals_hat,
                                   const __half* gamma,
                                   const __half* betta,
                                   const __half* vars,
                                   __half* inp_grad,
                                   bool invertible,
                                   int row_stride)
{
#ifdef HALF_PRECISION_AVAILABLE
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    int warp_num = iteration_stride >> WARP_SIZE_BITS;
    __shared__ float partialSum[MAX_WARP_NUM];

    __half2 vals_arr[NORM_REG];
    float2 vals_arr_f[NORM_REG];
    __half2 vals_hat_arr[NORM_REG];

    __half2* inp_grad_h = reinterpret_cast<__half2*>(inp_grad);
    const __half2* out_grad_h = reinterpret_cast<const __half2*>(out_grad);
    const __half2* vals_hat_h = reinterpret_cast<const __half2*>(vals_hat);

    inp_grad_h += (row * row_stride);
    out_grad_h += (row * row_stride);
    vals_hat_h += (row * row_stride);

    const __half2* gamma_h = reinterpret_cast<const __half2*>(gamma);
    const __half2* betta_h = (invertible ? reinterpret_cast<const __half2*>(betta) : nullptr);
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        __half2 gamma_reg = gamma_h[i * iteration_stride + id];
        vals_arr[i] = out_grad_h[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
        vals_hat_arr[i] =
            (invertible
                 ? (vals_hat_h[i * iteration_stride + id] - betta_h[i * iteration_stride + id]) /
                       gamma_reg
                 : vals_hat_h[i * iteration_stride + id]);
    }
    if ((high_index) < row_stride) {
        __half2 gamma_reg = gamma_h[high_index];
        vals_arr[iterations] = out_grad_h[high_index];
        vals_arr[iterations] *= gamma_reg;
        vals_hat_arr[iterations] =
            (invertible ? (vals_hat_h[high_index] - betta_h[high_index]) / gamma_reg
                        : vals_hat_h[high_index]);
        iterations++;
    }
    __half var_h = vars[row];
    __half2 var_reg = __halves2half2(var_h, var_h);

    float sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        __half2 result_h = (vals_hat_arr[i] * vals_arr[i] * h2sqrt(var_reg));
        float2 result_f = __half22float2(result_h);
        sum += result_f.x;
        sum += result_f.y;
        vals_arr[i] *= h2rsqrt(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);
    __half2 sum_h = __float2half2_rn(sum);

    for (int i = 0; i < iterations; i++) {
        __half2 temp = ((-sum_h * vals_hat_arr[i]) / (var_reg));
        vals_arr_f[i] = __half22float2(vals_arr[i]);
        float2 temp_f = __half22float2(temp);
        vals_arr_f[i].x += temp_f.x;
        vals_arr_f[i].y += temp_f.y;
    }
    sum = 0.f;

    for (int i = 0; i < iterations; i++) {
        sum += (vals_arr_f[i].x);
        sum += (vals_arr_f[i].y);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        vals_arr_f[i].x -= sum;
        vals_arr_f[i].y -= sum;
        __half2 temp = __float22half2_rn(vals_arr_f[i]);

        inp_grad_h[i * iteration_stride + id] = temp;
    }
    if ((high_index) < row_stride) {
        vals_arr_f[iterations].x -= sum;
        vals_arr_f[iterations].y -= sum;
        __half2 temp = __float22half2_rn(vals_arr_f[iterations]);

        inp_grad_h[high_index] = temp;
    }
#endif
}

template <>
void launch_layerNorm_backward<float>(const float* out_grad,
                                      const float* vals_hat,
                                      const float* vars,
                                      const float* gamma,
                                      float* gamma_grad,
                                      float* betta_grad,
                                      float* inp_grad,
                                      int batch,
                                      int hidden_dim,
                                      hipStream_t stream[2],
                                      bool invertible,
                                      const float* betta)
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<float><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad, vals_hat, gamma, betta, gamma_grad, betta_grad, batch, hidden_dim, invertible);

    dim3 grid_dim2(batch);

    if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 1;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 2;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads);

    LayerNormBackward2<<<grid_dim2, block_dim2, 0, stream[1]>>>(
        out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible, hidden_dim);
}

template <>
void launch_layerNorm_backward<__half>(const __half* out_grad,
                                       const __half* vals_hat,
                                       const __half* vars,
                                       const __half* gamma,
                                       __half* gamma_grad,
                                       __half* betta_grad,
                                       __half* inp_grad,
                                       int batch,
                                       int hidden_dim,
                                       hipStream_t stream[2],
                                       bool invertible,
                                       const __half* betta)
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    // LayerNormBackward1<__half><<<grid_dim, block_dim, 0, stream[0]>>>(
    //    out_grad, vals_hat, gamma, betta, gamma_grad, betta_grad, batch, hidden_dim, invertible);

    dim3 grid_dim2(batch);

    if (hidden_dim > 8192 && hidden_dim <= 16384)
        threads <<= 1;
    else if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 2;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 3;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads / 2);

    LayerNormBackward2<<<grid_dim2, block_dim2, 0, stream[1]>>>(
        out_grad, vals_hat, gamma, betta, vars, inp_grad, invertible, hidden_dim / 2);
}

/* Backward Normalize (Input-Gradient)
 * Using the means and variances from the input
 * This type of backward is not invertible!
 * We do the backward using the input (X)
 */

__global__ void LayerNormBackward2(const float* out_grad,
                                   const float* X_vals,
                                   const float* gamma,
                                   const float* vars,
                                   const float* means,
                                   float* inp_grad,
                                   int row_stride)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id >> WARP_SIZE_BITS;
    int warp_num = iteration_stride >> WARP_SIZE_BITS;
    __shared__ float partialSum[MAX_WARP_NUM];

    out_grad += (row * row_stride);
    X_vals += (row * row_stride);
    inp_grad += (row * row_stride);

    float vals_arr[NORM_REG];
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        float gamma_reg = gamma[i * iteration_stride + id];
        vals_arr[i] = out_grad[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
    }
    if ((high_index) < row_stride) {
        float gamma_reg = gamma[high_index];
        vals_arr[iterations] = out_grad[high_index];
        vals_arr[iterations] *= gamma_reg;
        iterations++;
    }

    float var_reg = vars[row];
    float mean_reg = means[row];

    float sum = 0;
    float xu[NORM_REG];
    for (int i = 0; i < iterations; i++) {
        xu[i] = (X_vals[i * iteration_stride + id] - mean_reg);
        sum += vals_arr[i] * xu[i];
        vals_arr[i] *= rsqrtf(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++) {
        vals_arr[i] += (-sum * xu[i] * rsqrtf(var_reg) / (var_reg));
    }

    sum = 0;
    for (int i = 0; i < iterations; i++) { sum += vals_arr[i]; }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);
    sum = g.shfl(sum, 0);
    sum /= row_stride;

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) inp_grad[i * iteration_stride + id] = (vals_arr[i] - sum);
    if ((high_index) < row_stride) inp_grad[high_index] = (vals_arr[iterations] - sum);
}

__global__ void LayerNormBackward2(const __half* out_grad,
                                   const __half* X_vals,
                                   const __half* gamma,
                                   const __half* vars,
                                   const __half* means,
                                   __half* inp_grad,
                                   int row_stride)
{
#ifdef HALF_PRECISION_AVAILABLE
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id >> WARP_SIZE_BITS;
    int warp_num = iteration_stride >> WARP_SIZE_BITS;

    __shared__ float partialSum[MAX_WARP_NUM];

    __half2 vals_arr[NORM_REG];
    float2 vals_arr_f[NORM_REG];
    __half2 xu[NORM_REG];

    __half2* inp_grad_h = reinterpret_cast<__half2*>(inp_grad);
    const __half2* out_grad_h = reinterpret_cast<const __half2*>(out_grad);
    const __half2* vals_hat_h = reinterpret_cast<const __half2*>(X_vals);

    inp_grad_h += (row * row_stride);
    out_grad_h += (row * row_stride);
    vals_hat_h += (row * row_stride);

    const __half2* gamma_h = reinterpret_cast<const __half2*>(gamma);
    int high_index = iterations * iteration_stride + id;

    __half mean_h = means[row];
    __half2 mean_reg = __halves2half2(mean_h, mean_h);
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        __half2 gamma_reg = gamma_h[i * iteration_stride + id];
        vals_arr[i] = out_grad_h[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;  // out_grad * gamma
        xu[i] = (vals_hat_h[i * iteration_stride + id] - mean_reg);
    }
    if ((high_index) < row_stride) {
        __half2 gamma_reg = gamma_h[high_index];
        vals_arr[iterations] = out_grad_h[high_index];
        vals_arr[iterations] *= gamma_reg;  // out_grad * gamma
        xu[iterations] = (vals_hat_h[high_index] - mean_reg);
        iterations++;
    }
    __half var_h = vars[row];
    __half2 var_reg = __halves2half2(var_h, var_h);

    float sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        __half2 result_h = (xu[i] * vals_arr[i]);
        float2 result_f = __half22float2(result_h);
        sum += result_f.x;
        sum += result_f.y;
        vals_arr[i] *= h2rsqrt(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);
    __half2 sum_h = __float2half2_rn(sum);

    for (int i = 0; i < iterations; i++) {
        __half2 xu_grad = ((-sum_h * xu[i] * h2rsqrt(var_reg)) / (var_reg));
        vals_arr_f[i] = __half22float2(vals_arr[i]);
        float2 xu_grad_f = __half22float2(xu_grad);
        vals_arr_f[i].x += xu_grad_f.x;
        vals_arr_f[i].y += xu_grad_f.y;
    }

    sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        sum += (vals_arr_f[i].x);
        sum += (vals_arr_f[i].y);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        vals_arr_f[i].x -= sum;
        vals_arr_f[i].y -= sum;
        __half2 temp = __float22half2_rn(vals_arr_f[i]);
        inp_grad_h[i * iteration_stride + id] = temp;
    }
    if ((high_index) < row_stride) {
        vals_arr_f[iterations].x -= sum;
        vals_arr_f[iterations].y -= sum;
        __half2 temp = __float22half2_rn(vals_arr_f[iterations]);
        inp_grad_h[high_index] = temp;
    }
#endif
}

template <>
void launch_layerNorm_backward<float>(const float* out_grad,
                                      const float* X_data,
                                      const float* vars,
                                      const float* means,
                                      const float* gamma,
                                      float* gamma_grad,
                                      float* betta_grad,
                                      float* inp_grad,
                                      int batch,
                                      int hidden_dim,
                                      hipStream_t stream[2])
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<float><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad, X_data, vars, means, gamma_grad, betta_grad, batch, hidden_dim);

    dim3 grid_dim2(batch);

    if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 1;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 2;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads);
    LayerNormBackward2<<<grid_dim2, block_dim2, 0, stream[1]>>>(
        out_grad, X_data, gamma, vars, means, inp_grad, hidden_dim);
}

template <>
void launch_layerNorm_backward<__half>(const __half* out_grad,
                                       const __half* X_data,
                                       const __half* vars,
                                       const __half* means,
                                       const __half* gamma,
                                       __half* gamma_grad,
                                       __half* betta_grad,
                                       __half* inp_grad,
                                       int batch,
                                       int hidden_dim,
                                       hipStream_t stream[2])
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<__half><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad, X_data, vars, means, gamma_grad, betta_grad, batch, hidden_dim);

    dim3 grid_dim2(batch);

    if (hidden_dim > 8192 && hidden_dim <= 16384)
        threads <<= 1;
    else if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 2;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 3;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads / 2);
    LayerNormBackward2<<<grid_dim2, block_dim2, 0, stream[1]>>>(
        out_grad, X_data, gamma, vars, means, inp_grad, hidden_dim / 2);
}

template <typename T>
__global__ void LayerNormBackward1_fused_add(const T* __restrict__ out_grad1,
                                             const T* __restrict__ out_grad2,
                                             const T* __restrict__ vals_hat,
                                             const T* __restrict__ gamma,
                                             const T* __restrict__ betta,
                                             T* __restrict__ gamma_grad,
                                             T* __restrict__ betta_grad,
                                             int rows,
                                             int width,
                                             bool invertible)
{
    __shared__ float betta_buffer[TILE_DIM][TILE_DIM + 1];
    __shared__ float gamma_buffer[TILE_DIM][TILE_DIM + 1];

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = threadIdx.y * width + idx;
    int y_stride = width * TILE_DIM;

    float betta_reg = (invertible ? (float)betta[idx] : 0.0f);
    float gamma_reg = (float)gamma[idx];

    // Loop across matrix height
    float betta_tmp = 0;
    float gamma_tmp = 0;
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        float grad = (float)out_grad1[offset] + (float)out_grad2[offset];
        float val = (invertible ? ((float)vals_hat[offset] - betta_reg) / gamma_reg
                                : (float)vals_hat[offset]);
        betta_tmp += grad;
        gamma_tmp += (val * grad);

        offset += y_stride;
    }

    betta_buffer[threadIdx.x][threadIdx.y] = betta_tmp;
    gamma_buffer[threadIdx.x][threadIdx.y] = gamma_tmp;

    __syncthreads();

    // Sum the shared buffer.
    float s1 = betta_buffer[threadIdx.y][threadIdx.x];
    float s2 = gamma_buffer[threadIdx.y][threadIdx.x];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < TILE_DIM; i <<= 1) {
        s1 += g.shfl_down(s1, i);
        s2 += g.shfl_down(s2, i);
    }

    if (threadIdx.x == 0) {
        int pos = blockIdx.x * TILE_DIM + threadIdx.y;
        betta_grad[pos] = s1;
        gamma_grad[pos] = s2;
    }
}

template <typename T>
__global__ void LayerNormBackward1_fused_add(const T* __restrict__ out_grad1,
                                             const T* __restrict__ out_grad2,
                                             const T* __restrict__ X_data,
                                             const T* __restrict__ vars,
                                             const T* __restrict__ means,
                                             T* __restrict__ gamma_grad,
                                             T* __restrict__ betta_grad,
                                             int rows,
                                             int width)
{
    __shared__ float betta_buffer[TILE_DIM][TILE_DIM + 1];
    __shared__ float gamma_buffer[TILE_DIM][TILE_DIM + 1];

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int offset = threadIdx.y * width + idx;
    int y_stride = width * TILE_DIM;

    int pos = blockIdx.x * TILE_DIM + threadIdx.y;
    // Loop across matrix height

    float betta_tmp = 0;
    float gamma_tmp = 0;
    for (int r = threadIdx.y; r < rows; r += TILE_DIM) {
        float grad = (float)out_grad1[offset] + (float)out_grad2[offset];
        float val = (float)X_data[offset];
        val = (val - (float)means[r]) * rsqrtf((float)vars[r]);
        betta_tmp += grad;
        gamma_tmp += (val * grad);

        offset += y_stride;
    }

    betta_buffer[threadIdx.x][threadIdx.y] = betta_tmp;
    gamma_buffer[threadIdx.x][threadIdx.y] = gamma_tmp;

    __syncthreads();

    // Sum the shared buffer.
    float s1 = betta_buffer[threadIdx.y][threadIdx.x];
    float s2 = gamma_buffer[threadIdx.y][threadIdx.x];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < TILE_DIM; i <<= 1) {
        s1 += g.shfl_down(s1, i);
        s2 += g.shfl_down(s2, i);
    }

    if (threadIdx.x == 0) {
        betta_grad[pos] = s1;
        gamma_grad[pos] = s2;
    }
}

__global__ void LayerNormBackward2_fused_add(const float* out_grad1,
                                             const float* out_grad2,
                                             const float* vals_hat,
                                             const float* gamma,
                                             const float* betta,
                                             const float* vars,
                                             float* inp_grad,
                                             bool invertible,
                                             int row_stride)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    int warp_num = iteration_stride >> WARP_SIZE_BITS;
    __shared__ float partialSum[MAX_WARP_NUM];

    out_grad1 += (row * row_stride);
    out_grad2 += (row * row_stride);
    vals_hat += (row * row_stride);
    inp_grad += (row * row_stride);

    float vals_arr[NORM_REG];
    float vals_hat_arr[NORM_REG];
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        float gamma_reg = gamma[i * iteration_stride + id];
        vals_arr[i] = out_grad1[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
        vals_hat_arr[i] =
            (invertible ? (vals_hat[i * iteration_stride + id] - betta[i * iteration_stride + id]) /
                              gamma_reg
                        : vals_hat[i * iteration_stride + id]);
    }
    if ((high_index) < row_stride) {
        float gamma_reg = gamma[high_index];
        vals_arr[iterations] = out_grad1[high_index];
        vals_arr[iterations] *= gamma_reg;
        vals_hat_arr[iterations] =
            (invertible ? (vals_hat[high_index] - betta[high_index]) / gamma_reg
                        : vals_hat[high_index]);
        iterations++;
    }

    float var_reg = vars[row];

    float sum = 0;
    for (int i = 0; i < iterations; i++) {
        sum += vals_hat_arr[i] * vals_arr[i] * sqrtf(var_reg);
        vals_arr[i] *= rsqrtf(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++) { vals_arr[i] += ((-sum * vals_hat_arr[i]) / var_reg); }

    sum = 0;
    for (int i = 0; i < iterations; i++) { sum += vals_arr[i]; }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);
    sum = g.shfl(sum, 0);
    sum /= row_stride;

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++)
        inp_grad[i * iteration_stride + id] =
            (vals_arr[i] - sum) + out_grad2[i * iteration_stride + id];
    if ((high_index) < row_stride)
        inp_grad[high_index] = (vals_arr[iterations] - sum) + out_grad2[high_index];
}

__global__ void LayerNormBackward2_fused_add(const __half* out_grad1,
                                             const __half* out_grad2,
                                             const __half* vals_hat,
                                             const __half* gamma,
                                             const __half* betta,
                                             const __half* vars,
                                             __half* inp_grad,
                                             bool invertible,
                                             int row_stride)
{
#ifdef HALF_PRECISION_AVAILABLE
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    int warp_num = iteration_stride >> WARP_SIZE_BITS;
    __shared__ float partialSum[MAX_WARP_NUM];

    __half2 vals_arr[NORM_REG];
    float2 vals_arr_f[NORM_REG];
    __half2 vals_hat_arr[NORM_REG];

    // float2 result[iterations];

    __half2* inp_grad_h = reinterpret_cast<__half2*>(inp_grad);
    const __half2* out_grad_h1 = reinterpret_cast<const __half2*>(out_grad1);
    const __half2* out_grad_h2 = reinterpret_cast<const __half2*>(out_grad2);
    const __half2* vals_hat_h = reinterpret_cast<const __half2*>(vals_hat);

    inp_grad_h += (row * row_stride);
    out_grad_h1 += (row * row_stride);
    out_grad_h2 += (row * row_stride);
    vals_hat_h += (row * row_stride);

    const __half2* gamma_h = reinterpret_cast<const __half2*>(gamma);
    const __half2* betta_h = (invertible ? reinterpret_cast<const __half2*>(betta) : nullptr);
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        __half2 gamma_reg = gamma_h[i * iteration_stride + id];
        vals_arr[i] = out_grad_h1[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;  // out_grad * gamma
        vals_hat_arr[i] =
            (invertible
                 ? (vals_hat_h[i * iteration_stride + id] - betta_h[i * iteration_stride + id]) /
                       gamma_reg
                 : vals_hat_h[i * iteration_stride + id]);
    }
    if ((high_index) < row_stride) {
        __half2 gamma_reg = gamma_h[high_index];
        vals_arr[iterations] = out_grad_h1[high_index];
        vals_arr[iterations] *= gamma_reg;  // out_grad * gamma
        vals_hat_arr[iterations] =
            (invertible ? (vals_hat_h[high_index] - betta_h[high_index]) / gamma_reg
                        : vals_hat_h[high_index]);
        iterations++;
    }
    __half var_h = vars[row];
    __half2 var_reg = __halves2half2(var_h, var_h);

    float sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        __half2 result_h = (vals_hat_arr[i] * vals_arr[i] * h2sqrt(var_reg));
        float2 result_f = __half22float2(result_h);
        sum += result_f.x;
        sum += result_f.y;
        vals_arr[i] *= h2rsqrt(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);
    __half2 sum_h = __float2half2_rn(sum);

    for (int i = 0; i < iterations; i++) {
        __half2 temp = ((-sum_h * vals_hat_arr[i]) / (var_reg));
        vals_arr_f[i] = __half22float2(vals_arr[i]);
        float2 temp_f = __half22float2(temp);
        vals_arr_f[i].x += temp_f.x;
        vals_arr_f[i].y += temp_f.y;
    }
    sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        sum += (vals_arr_f[i].x);
        sum += (vals_arr_f[i].y);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        vals_arr_f[i].x -= sum;
        vals_arr_f[i].y -= sum;
        __half2 temp = __float22half2_rn(vals_arr_f[i]);

        inp_grad_h[i * iteration_stride + id] = temp + out_grad_h2[i * iteration_stride + id];
    }
    if ((high_index) < row_stride) {
        vals_arr_f[iterations].x -= sum;
        vals_arr_f[iterations].y -= sum;
        __half2 temp = __float22half2_rn(vals_arr_f[iterations]);

        inp_grad_h[high_index] = temp + out_grad_h2[high_index];
    }
#endif
}

template <>
void launch_layerNorm_backward_fused_add<float>(const float* out_grad1,
                                                const float* out_grad2,
                                                const float* vals_hat,
                                                const float* vars,
                                                const float* gamma,
                                                float* gamma_grad,
                                                float* betta_grad,
                                                float* inp_grad,
                                                int batch,
                                                int hidden_dim,
                                                hipStream_t stream[2],
                                                bool invertible,
                                                const float* betta)
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);
    LayerNormBackward1<float><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad1, vals_hat, gamma, betta, gamma_grad, betta_grad, batch, hidden_dim, invertible);

    dim3 grid_dim2(batch);

    if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 1;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 2;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads);
    LayerNormBackward2_fused_add<<<grid_dim2, block_dim2, 0, stream[1]>>>(
        out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible, hidden_dim);
}

template <>
void launch_layerNorm_backward_fused_add<__half>(const __half* out_grad1,
                                                 const __half* out_grad2,
                                                 const __half* vals_hat,
                                                 const __half* vars,
                                                 const __half* gamma,
                                                 __half* gamma_grad,
                                                 __half* betta_grad,
                                                 __half* inp_grad,
                                                 int batch,
                                                 int hidden_dim,
                                                 hipStream_t stream[2],
                                                 bool invertible,
                                                 const __half* betta)
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<__half><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad1, vals_hat, gamma, betta, gamma_grad, betta_grad, batch, hidden_dim, invertible);

    dim3 grid_dim2(batch);

    if (hidden_dim > 8192 && hidden_dim <= 16384)
        threads <<= 1;
    else if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 2;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 3;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads / 2);
    LayerNormBackward2_fused_add<<<grid_dim2, block_dim2, 0, stream[1]>>>(
        out_grad1, out_grad2, vals_hat, gamma, betta, vars, inp_grad, invertible, hidden_dim / 2);
}

/* Backward Normalize (Input-Gradient)
 * Using the means and variances from the input
 * This type of backward is not invertible!
 * We do the backward using the input (X)
 */

__global__ void LayerNormBackward2_fused_add(const float* out_grad1,
                                             const float* out_grad2,
                                             const float* X_vals,
                                             const float* gamma,
                                             const float* vars,
                                             const float* means,
                                             float* inp_grad,
                                             int row_stride)
{
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    int warp_num = iteration_stride >> WARP_SIZE_BITS;
    __shared__ float partialSum[MAX_WARP_NUM];

    float vals_arr[NORM_REG];
    float vals_hat_arr[NORM_REG];

    out_grad1 += (row * row_stride);
    out_grad2 += (row * row_stride);
    X_vals += (row * row_stride);
    inp_grad += (row * row_stride);
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        float gamma_reg = gamma[i * iteration_stride + id];
        vals_arr[i] = out_grad1[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;
        vals_hat_arr[i] = X_vals[i * iteration_stride + id];
    }
    if ((high_index) < row_stride) {
        float gamma_reg = gamma[high_index];
        vals_arr[iterations] = out_grad1[high_index];
        vals_arr[iterations] *= gamma_reg;
        vals_hat_arr[iterations] = X_vals[high_index];
        iterations++;
    }

    float var_reg = vars[row];
    float mean_reg = means[row];

    float sum = 0;
    float xu[NORM_REG];
    for (int i = 0; i < iterations; i++) {
        xu[i] = (vals_hat_arr[i] - mean_reg);
        sum += vals_arr[i] * xu[i];
        vals_arr[i] *= rsqrtf(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= row_stride;

    for (int i = 0; i < iterations; i++) {
        vals_arr[i] += (-sum * xu[i] * rsqrtf(var_reg) / (var_reg));
    }

    sum = 0;
    for (int i = 0; i < iterations; i++) { sum += vals_arr[i]; }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);
    sum = g.shfl(sum, 0);
    sum /= row_stride;

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++)
        inp_grad[i * iteration_stride + id] =
            (vals_arr[i] - sum) + out_grad2[i * iteration_stride + id];
    if ((high_index) < row_stride)
        inp_grad[high_index] = (vals_arr[iterations] - sum) + out_grad2[high_index];
}

__global__ void LayerNormBackward2_fused_add(const __half* out_grad1,
                                             const __half* out_grad2,
                                             const __half* X_vals,
                                             const __half* gamma,
                                             const __half* vars,
                                             const __half* means,
                                             __half* inp_grad,
                                             int row_stride)
{
#ifdef HALF_PRECISION_AVAILABLE
    int iteration_stride = blockDim.x;
    int iterations = row_stride / iteration_stride;

    cg::thread_block b = cg::this_thread_block();
    cg::thread_block_tile<WARP_SIZE> g = cg::tiled_partition<WARP_SIZE>(b);

    int row = blockIdx.x;
    int id = threadIdx.x;
    int wid = id / WARP_SIZE;
    int warp_num = iteration_stride >> WARP_SIZE_BITS;

    __shared__ float partialSum[MAX_WARP_NUM];

    __half2 vals_arr[NORM_REG];
    float2 vals_arr_f[NORM_REG];
    __half2 vals_hat_arr[NORM_REG];

    __half2* inp_grad_h = reinterpret_cast<__half2*>(inp_grad);
    const __half2* out_grad_h1 = reinterpret_cast<const __half2*>(out_grad1);
    const __half2* out_grad_h2 = reinterpret_cast<const __half2*>(out_grad2);
    const __half2* vals_hat_h = reinterpret_cast<const __half2*>(X_vals);

    out_grad_h1 += (row * row_stride);
    out_grad_h2 += (row * row_stride);
    inp_grad_h += (row * row_stride);
    vals_hat_h += (row * row_stride);

    const __half2* gamma_h = reinterpret_cast<const __half2*>(gamma);
    int high_index = iterations * iteration_stride + id;
#pragma unroll
    for (int i = 0; i < iterations; i++) {
        __half2 gamma_reg = gamma_h[i * iteration_stride + id];
        vals_arr[i] = out_grad_h1[i * iteration_stride + id];
        vals_arr[i] *= gamma_reg;  // out_grad * gamma
        vals_hat_arr[i] = vals_hat_h[i * iteration_stride + id];
    }
    if ((high_index) < row_stride) {
        __half2 gamma_reg = gamma_h[high_index];
        vals_arr[iterations] = out_grad_h1[high_index];
        vals_arr[iterations] *= gamma_reg;  // out_grad * gamma
        vals_hat_arr[iterations] = vals_hat_h[high_index];
        iterations++;
    }

    __half mean_h = means[row];
    __half var_h = vars[row];
    __half2 var_reg = __halves2half2(var_h, var_h);
    __half2 mean_reg = __halves2half2(mean_h, mean_h);
    __half2 xu[NORM_REG];

    float sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        xu[i] = (vals_hat_arr[i] - mean_reg);
        __half2 result_h = (xu[i] * vals_arr[i]);
        float2 result_f = __half22float2(result_h);
        sum += result_f.x;
        sum += result_f.y;
        vals_arr[i] *= h2rsqrt(var_reg);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);
    __half2 sum_h = __float2half2_rn(sum);

    for (int i = 0; i < iterations; i++) {
        __half2 xu_grad = ((-sum_h * xu[i] * h2rsqrt(var_reg)) / (var_reg));
        vals_arr_f[i] = __half22float2(vals_arr[i]);
        float2 xu_grad_f = __half22float2(xu_grad);
        vals_arr_f[i].x += xu_grad_f.x;
        vals_arr_f[i].y += xu_grad_f.y;
    }

    sum = 0.f;
    for (int i = 0; i < iterations; i++) {
        sum += (vals_arr_f[i].x);
        sum += (vals_arr_f[i].y);
    }

    for (int i = 1; i < WARP_SIZE; i *= 2) { sum += g.shfl_down(sum, i); }

    if (g.thread_rank() == 0) partialSum[wid] = sum;

    __syncthreads();

    if (g.thread_rank() < warp_num) sum = partialSum[g.thread_rank()];

#ifndef __STOCHASTIC_MODE__
    __syncthreads();
#endif

    for (int i = 1; i < warp_num; i *= 2) sum += g.shfl_down(sum, i);

    sum = g.shfl(sum, 0);
    sum /= (2 * row_stride);

    iterations = row_stride / iteration_stride;
    for (int i = 0; i < iterations; i++) {
        vals_arr_f[i].x -= sum;
        vals_arr_f[i].y -= sum;
        __half2 temp = __float22half2_rn(vals_arr_f[i]);
        inp_grad_h[i * iteration_stride + id] = temp + out_grad_h2[i * iteration_stride + id];
    }
    if ((high_index) < row_stride) {
        vals_arr_f[iterations].x -= sum;
        vals_arr_f[iterations].y -= sum;
        __half2 temp = __float22half2_rn(vals_arr_f[iterations]);
        inp_grad_h[high_index] = temp + out_grad_h2[high_index];
    }
#endif
}

template <>
void launch_layerNorm_backward_fused_add<float>(const float* out_grad1,
                                                const float* out_grad2,
                                                const float* X_data,
                                                const float* vars,
                                                const float* means,
                                                const float* gamma,
                                                float* gamma_grad,
                                                float* betta_grad,
                                                float* inp_grad,
                                                int batch,
                                                int hidden_dim,
                                                hipStream_t stream[2])
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<float><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad1, X_data, vars, means, gamma_grad, betta_grad, batch, hidden_dim);

    dim3 grid_dim2(batch);

    if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 1;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 2;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads);
    LayerNormBackward2_fused_add<<<grid_dim2, block_dim2, 0, stream[1]>>>(
        out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad, hidden_dim);
}

template <>
void launch_layerNorm_backward_fused_add<__half>(const __half* out_grad1,
                                                 const __half* out_grad2,
                                                 const __half* X_data,
                                                 const __half* vars,
                                                 const __half* means,
                                                 const __half* gamma,
                                                 __half* gamma_grad,
                                                 __half* betta_grad,
                                                 __half* inp_grad,
                                                 int batch,
                                                 int hidden_dim,
                                                 hipStream_t stream[2])
{
    int threads = THREADS;

    dim3 grid_dim(hidden_dim / TILE_DIM);
    dim3 block_dim(TILE_DIM, TILE_DIM);

    LayerNormBackward1<__half><<<grid_dim, block_dim, 0, stream[0]>>>(
        out_grad1, X_data, vars, means, gamma_grad, betta_grad, batch, hidden_dim);

    dim3 grid_dim2(batch);

    if (hidden_dim > 8192 && hidden_dim <= 16384)
        threads <<= 1;
    else if (hidden_dim > 16384 && hidden_dim <= 32768)
        threads <<= 2;
    else if (hidden_dim > 32768 && hidden_dim <= 65536)
        threads <<= 3;
    else if (hidden_dim > 65536)
        throw std::runtime_error("Unsupport hidden_dim.");

    dim3 block_dim2(threads / 2);
    LayerNormBackward2_fused_add<<<grid_dim2, block_dim2, 0, stream[1]>>>(
        out_grad1, out_grad2, X_data, gamma, vars, means, inp_grad, hidden_dim / 2);
}
